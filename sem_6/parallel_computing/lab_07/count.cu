#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void count_word_occurrences(char *sentence, char *word,
				       int sentence_length, int word_length,
				       int *count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = idx; i < sentence_length - word_length + 1; i += stride) {
		bool match = true;
		for (int j = 0; j < word_length; j++) {
			if (sentence[i + j] != word[j]) {
				match = false;
				break;
			}
		}
		if (match)
			atomicAdd(count, 1);
	}
}

int main(void)
{
	const char *sentence = "The quick brown fox jumps over the lazy dog. The fox is quick.";
	const char *word = "the";
	int sentence_length = strlen(sentence);
	int word_length = strlen(word);

	char *d_sentence, *d_word;
	int *d_count, h_count = 0;

	hipMalloc(&d_sentence, sentence_length * sizeof(char));
	hipMalloc(&d_word, word_length * sizeof(char));
	hipMalloc(&d_count, sizeof(int));

	hipMemcpy(d_sentence, sentence, sentence_length * sizeof(char),
		   hipMemcpyHostToDevice);
	hipMemcpy(d_word, word, word_length * sizeof(char),
		   hipMemcpyHostToDevice);
	hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

	int block_size = 256;
	int num_blocks = (sentence_length + block_size - 1) / block_size;

	count_word_occurrences<<<num_blocks, block_size>>>(d_sentence, d_word,
							   sentence_length,
							   word_length, d_count);

	hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("The word '%s' appears %d times in the sentence.\n", word, h_count);

	hipFree(d_sentence);
	hipFree(d_word);
	hipFree(d_count);

	return 0;
}
